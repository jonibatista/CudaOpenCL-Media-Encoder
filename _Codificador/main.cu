#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
///   Copyright (C) 2008 by Nelson Carreira Francisco                        ///
///   eng.nelsito@gmail.com                                                  ///
///                                                                          ///
///   This program is free software; you can redistribute it and/or modify   ///
///   it under the terms of the GNU General Public License as published by   ///
///   the Free Software Foundation; either version 2 of the License, or      ///
///   (at your option) any later version.                                    ///
///                                                                          ///
///   This program is distributed in the hope that it will be useful,        ///
///   but WITHOUT ANY WARRANTY; without even the implied warranty of         ///
///   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the          ///
///   GNU General Public License for more details.                           ///
///                                                                          ///
///   You should have received a copy of the GNU General Public License      ///
///   along with this program; if not, write to the                          ///
///   Free Software Foundation, Inc.,                                        ///
///   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.              ///
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
///   Implementacaoo de um codificador de imagens baseado em                 ///
///   Quantificacao vectorial                                                ///
///   Nelson Carreira Francisco                                              ///
////////////////////////////////////////////////////////////////////////////////


#ifdef HAVE_CONFIG_H
#include <config.h>
#endif


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <unistd.h>
#include <ctype.h>
#include <time.h>
#include <iostream>

#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include "params.h"


////////////////////////////////////////////////////////////////////////////////
///                         CONSTANTS ERROR                                  ///
////////////////////////////////////////////////////////////////////////////////
#define ERROR_INVALID_PARAMETERS 1
#define ERROR_ALLOCATE_MEMORY 2
#define ERROR_OPEN_FILE
#define ERROR_INVALID_PARAMETERS 1



////////////////////////////////////////////////////////////////////////////////
///                              CONSTANTS                                   ///
////////////////////////////////////////////////////////////////////////////////
#define RANGE_LUMINANCE 255	// Range of image luminance values
#define PERMS       	0644	// File acess permits:RW for the users and R for the others
#define RANGEY      	 255	// Range level of luminance

#define Clip1(a)            ((a)>255?255:((a)<0?0:(a)))
/**
 * <p> Function to process CUDA errors </p>
 *
 * @param err [IN] CUDA error to process (usually the code returned by the cuda function)
 * @param line [IN] line of source code where function is called
 * @param file [IN] name of source file where function is called
 * @return on error, the function terminates the process with EXIT_FAILURE code.
 *
 * source: "CUDA by Example: An Introduction to General-Purpose "
 * GPU Programming", Jason Sanders, Edward Kandrot, NVIDIA, July 2010
 * @note: the function should be called through the macro 'HANDLE_ERROR'
 **/
static void
HandleError (hipError_t err, const char *file, int line)
{
  if (err != hipSuccess)
    {
      printf ("[ERROR] '%s' (%d) in '%s' at line '%d'\n",
	      hipGetErrorString (err), err, file, line);
      exit (EXIT_FAILURE);
    }
}

/**
 * <p>HANDLE_ERROR macro.</p>
 *
 * Wrapping macro for HandleError function (provides "file" and "line" parameters).
 *
 * @param err [IN] CUDA error
 * @return on error, the calling process is terminated
 **/
#define HANDLE_ERROR(err) (HandleError((err), __FILE__, __LINE__ ))
const int G_ThreadsPerBlock = 512;	//MAX_T;;
const int G_BlocksPerGrid = 65536;	//

////////////////////////////////////////////////////////////////////////////////
///                             CUDA KERNEL                                  ///
////////////////////////////////////////////////////////////////////////////////
/**
 * <p>calculate all quad error and create the pgm encoded</p>
 * 
 * @param num_codewords number of rows of the dictionary
 * @param block_size size of the block (horizontal * verticar coordinates)
 * @param dev_dict ditionary data in memory
 * @param dev_pgm pgm image in memory
 * @param dev_pgm_coded encoding result
 */
__global__ void
encoding_pgm (int num_codewords, int pgm_block_size, int *dev_dict,
	      int *dev_pgm, int *dev_pgm_coded)
{
  __shared__ float cache[G_ThreadsPerBlock];

  int i;
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  int cache_index = threadIdx.x;
  float temp = 0.0;

  while (tid < num_codewords)
    {
      i = 0;
      temp = 0.0;

      int idx_dict = threadIdx.x * pgm_block_size;
      int idx_block = 0;

      while (i < pgm_block_size)
	{
	  idx_block = (blockIdx.x * blockDim.x) + i;
	  temp +=
	    ((dev_dict[idx_dict + i] -
	      dev_pgm[idx_block + i]) * (dev_dict[idx_dict + i] -
					 dev_pgm[idx_block + i]));
	  i++;
	}

      cache[cache_index] = temp;
      __syncthreads ();
    }

  if (threadIdx.x == 0)
    {
      float aux = FLT_MAX;

      for (i = 0; i < blockDim.x; i++)
	{
	  if (cache[i] < aux)
	    {
	      // printf(" %d ", i);
	      aux = cache[i];
	      dev_pgm_coded[blockIdx.x] = i;
	    }
	}
    }
}

////////////////////////////////////////////////////////////////////////////////
///                           PROTOTYPES DEFINITION                          ///
////////////////////////////////////////////////////////////////////////////////

void read_header_pgm (int *ysize, int *xsize, char *file_name);
void read_file_pgm (int **pelimg, int *ysize, int *xsize, char *file_name);
void v_read_file_pgm (int *pelimg, int *ysize, int *xsize, char *file_name);
int **int_matrix (int nr, int nc);
float **floatmatrix (int nr, int nc);
int *int_vector (int nr, int nc);
void sort_pgm_blocks (int *v_pgm, int *v_sort_pgm, int block_x, int block_y,
		      int xsize, int ysize);
float quad_err (int index_dic, int block_size, int *original_block);
void load_dictionary (char *file_name, int *num_codewords, int *block_size_x,
		      int *block_size_y);
double calculate_psnr (int **origblk, int **cmpblk, int nline, int npixel);
double calculate_mse (int **origblk, int **cmpblk, int nline, int npixel);
void write_index (int index, int bits_index, long *bits_count,
		  int *bits_to_go, int *buffer, FILE * pointf_out);

void write_f_pgm (int **im_matrix, int nline, int npixel, char *filename);
unsigned char **ucmatrix (int nrl, int nrh, int ncl, int nch);

//Funcoes de escrita de bits para o ficheiro
void output_bit (int bit, FILE * output_file, int *buffer, int *bits_to_go,
		 long *bits_count);
void done_outputing_bits (FILE * output_file, int *buffer, int *bits_to_go);
int max_mumber_threads ();


////////////////////////////////////////////////////////////////////////////////
///                               GLOBAL VARIABLES                           ///
////////////////////////////////////////////////////////////////////////////////
int *G_dic;


////////////////////////////////////////////////////////////////////////////////
///                                  FUNCTIONS                               ///
////////////////////////////////////////////////////////////////////////////////

int
main (int argc, char *argv[])
{
  int **image_orig, **image_out;
  int *v_pgm, *v_pgm_sorted, *v_pgm_coded;

  struct gengetopt_args_info args_info;

  int *original_block;

  int i, j, i1, j1, n;
  int index = -1;		/* Dummy value -- index is set before its usage */
  int average;
  float distortion;
  float aux = 0;
  double psnr, mse;

  clock_t start, end;
  double elapsed = 0;
  start = clock ();

  /* THE BIT BUFFER */
  int buffer = 0;		/* Bits buffered for output                 */
  int bits_to_go = 8;		/* Number of bits still in buffer               */
  long bits_count = 0;

  /*
   * The following variables are loaded from the dictionary  file
   **/
  int block_size = 0;
  int block_size_x = 0;
  int block_size_y = 0;
  int num_codewords = 0;

  int bits_index = 0;

  int ysize = 0, xsize = 0;	/* The dimensions of the original image */

  char *inname, *outname, *dic_name;	//Nome dos ficheiros de input e output

  FILE *pointf_out;



  // validate parameters
  if (cmdline_parser (argc, argv, &args_info) != 0)
    {
      exit (ERROR_INVALID_PARAMETERS);
    }

  inname = args_info.imagem_arg;
  dic_name = args_info.dicionario_arg;
  outname = args_info.ficheiro_arg;


  //Carrega dicionario
  load_dictionary (dic_name, &num_codewords, &block_size_x, &block_size_y);
  bits_index = ceil (log (num_codewords) / log (2));
  block_size = block_size_x * block_size_y;

  original_block = (int *) calloc (block_size, sizeof (int));
  if (!original_block)
    {
      printf ("int_matrix() - allocation failure 1 \n");
      exit (1);
    }

  //Le imagem a comprimir
  printf ("\n imagem a comprimir            : %s", inname);
  read_header_pgm (&ysize, &xsize, inname);	/* Reads the PGM file and returns the picture size */

  // load pgm to vector
  v_pgm = int_vector (ysize, xsize);
  v_read_file_pgm (v_pgm, &ysize, &xsize, inname);

  // create the vector that will contain the coded pgm
  v_pgm_coded = int_vector (ysize / block_size_y, xsize / block_size_x);

  // old stuff (using matrixes)
  image_orig = int_matrix (ysize, xsize);
  image_out = int_matrix (ysize, xsize);


  printf ("\n Tamanho (%dx%d)             : %d pixels", xsize, ysize,
	  xsize * ysize);

  for (i = 0; i < ysize; i++)
    {
      for (j = 0; j < xsize; j++)
	{
	  image_orig[i][j] = v_pgm[i * xsize + j];
	}
    }

  int num_blocks = (ysize / block_size_y) * (xsize / block_size_x);
  printf ("\n Total de blocos %dx%d na imagem : %d blocos",
	  block_size_y, block_size_x, num_blocks);
  printf ("\n-----------------------------------------------------");

  //Calcula a average das luminancias da imagem

  for (i = 0; i < ysize; i++)
    {
      for (j = 0; j < xsize; j++)
	{
	  aux += v_pgm[i * xsize + j];
	}
    }
  average = aux / ((xsize) * (ysize));
  printf ("\n average                         : %d", average);
  fflush (stdout);
  //-----------------------------------------


  //Subtrai a média a todos os pixels
  for (i = 0; i < ysize; i++)
    {
      for (j = 0; j < xsize; j++)
	{
	  v_pgm[i * xsize + j] -= average;
	}
    }
  //-----------------------------------------


  pointf_out = fopen (outname, "w");
  if (pointf_out == NULL)
    {
      fprintf (stderr, "Impossivel criar ficheiro de saida: %s\n\n", outname);
      exit (1);
    }

  fprintf (pointf_out, "%d\n", xsize);
  fprintf (pointf_out, "%d\n", ysize);
  fprintf (pointf_out, "%d\n", average);

  // sort the pixels of the each block
  v_pgm_sorted = int_vector (ysize, xsize);
  sort_pgm_blocks (v_pgm, v_pgm_sorted, block_size_x, block_size_y, xsize,
		   ysize);

  int *v_pgm_coded2;
  v_pgm_coded2 = int_vector (ysize / block_size_y, xsize / block_size_x);

  // calculate the quad error. (this will be executed on GPU)
  for (i = 0; i < ysize * xsize; i += (block_size))
    {
      for (j = 0; j < block_size; j++)
	{
	  original_block[j] = v_pgm_sorted[i + j];
	}
      distortion = FLT_MAX;
      for (n = 0; n < num_codewords; n++)
	{			//Varre todos os elementos do codebook
	  aux = quad_err (n, block_size, original_block);
	  if (aux < distortion)
	    {
	      index = n;
	      distortion = aux;
	    }
	}

      v_pgm_coded2[i / (block_size)] = index;
    }



  //
  // CUDA STUFF
  // calculate all quad error and create the pgm encoded
  //

  // the gpu device vectors
  int *dev_pgm;
  int *dev_dict;
  int *dev_pgm_coded;

  // alloc memory to cuda vectors
  HANDLE_ERROR (hipMalloc
		((void **) &dev_pgm,
		 (size_t) (ysize * xsize) * sizeof (int)));
  HANDLE_ERROR (hipMalloc
		((void **) &dev_dict,
		 (size_t) (ysize * xsize) * sizeof (int)));
  HANDLE_ERROR (hipMalloc
		((void **) &dev_pgm_coded, G_BlocksPerGrid * sizeof (int)));

  // copy the vectors data fom host to gpu device
  HANDLE_ERROR (hipMemcpy
		(dev_pgm, v_pgm, (ysize * xsize) * sizeof (int),
		 hipMemcpyHostToDevice));
  HANDLE_ERROR (hipMemcpy
		(dev_dict, G_dic, num_codewords * block_size * sizeof (int),
		 hipMemcpyHostToDevice));

  // execute GPU KERNEL
  encoding_pgm << <G_BlocksPerGrid, G_ThreadsPerBlock >> >(num_codewords,
							   block_size_x *
							   block_size_y,
							   dev_dict, dev_pgm,
							   dev_pgm_coded);

  // copy the vector with the pgm coded from dpu decive to host
  HANDLE_ERROR (hipMemcpy
		(v_pgm_coded, dev_pgm_coded, (G_BlocksPerGrid) * sizeof (int),
		 hipMemcpyDeviceToHost));

  // show result
  /*printf("-----------------------------------------------------------------------------------------------\n");
     for (i = 0; i < G_BlocksPerGrid; i++) {
     printf("%d", v_pgm_coded[i]);
     } */
  printf
    ("-----------------------------------------------------------------------------------------------\n");

  // cuda free memory
  hipFree (dev_pgm);
  dev_pgm = NULL;
  hipFree (dev_dict);
  dev_dict = NULL;
  hipFree (dev_pgm_coded);
  dev_pgm_coded = NULL;

  for (i = 0; i < G_BlocksPerGrid; i++)
    {
      printf ("%d=%d", v_pgm_coded2[i], v_pgm_coded2[i]);
    }


  // verificar esta código... 
  for (i = 0; i < ysize; i += block_size_y)
    {
      for (j = 0; j < xsize; j += block_size_x)
	{
	  for (i1 = 0; i1 < block_size_y; i1++)
	    {
	      for (j1 = 0; j1 < block_size_x; j1++)
		{
		  image_out[i + i1][j + j1] =
		    G_dic[index * block_size_x + (i1 * block_size_x + j1)];
		}
	    }

	}
    }

  // write the coded pgm to file
  for (i = 0; i < (ysize / block_size_y); i++)
    {
      for (j = 0; j < (xsize / block_size_x); j++)
	{
	  write_index (v_pgm_coded2[i * (xsize / block_size_x) + j],
		       bits_index, &bits_count, &bits_to_go, &buffer,
		       pointf_out);
	}
    }


  done_outputing_bits (pointf_out, &buffer, &bits_to_go);

  end = clock ();
  elapsed += ((double) (end - start)) / CLOCKS_PER_SEC;
  start = clock ();

  //Soma novamente a média a todos os pixels
  for (i = 0; i < ysize; i++)
    {

      for (j = 0; j < xsize; j++)
	{
	  image_out[i][j] += average;
	}
    }
  //-----------------------------------------


  psnr = calculate_psnr (image_orig, image_out, ysize, xsize);
  mse = calculate_mse (image_orig, image_out, ysize, xsize);
  printf ("\n Tempo total de execucao       : %9.3f segundos", elapsed);
  printf ("\n Total Bits                    : %ld bits (%ld Bytes)",
	  bits_count, bits_count / 8);

  float rate = (float) (bits_count) / (float) (xsize * ysize);

  printf ("\n Taxa                          : %.2f bits/pixel", rate);
  printf ("\n psnr                          : %f dB", psnr);
  printf ("\n mse                           : %f", mse);
  printf ("\n-----------------------------------------------------\n\n");


  //write_f_pgm(image_out, *ysize, *xsize, "Testeout.pgm");

  fclose (pointf_out);

  // now free the memory
  for (i = 0; i < ysize; i++)
    {
      free (image_orig[i]);
      free (image_out[i]);
    }

  //free memory
  free (image_orig);
  image_orig = NULL;
  free (image_out);
  image_out = NULL;
  free (v_pgm_coded);
  v_pgm_coded = NULL;
  free (v_pgm);
  v_pgm = NULL;
  free (v_pgm_sorted);
  v_pgm_sorted = NULL;
  free (G_dic);
  G_dic = NULL;

  return EXIT_SUCCESS;
}

/**
 * <p> Gets the max number of threads for this device </p>
 *
 * @return max number of threads for this device
 */
int
max_number_threads ()
{
  hipDeviceProp_t prop;
  hipGetDeviceProperties (&prop, 0);
  return prop.maxThreadsPerBlock;
}

/**
 *
 * @param index
 * @param bits_index
 * @param bits_count
 * @param bits_to_go
 * @param buffer
 * @param pointf_out
 */
void
write_index (int index, int bits_index, long *bits_count, int *bits_to_go,
	     int *buffer, FILE * pointf_out)
{
  int k;
  int mask;
  int bit_to_write;

  mask = pow (2, bits_index - 1);
  for (k = 0; k < bits_index; k++)
    {
      bit_to_write = (mask & index) / mask;
      index = index << 1;
      output_bit (bit_to_write, pointf_out, buffer, bits_to_go, bits_count);
    }

}

/**
 * <p> Load the dictionary file to memory. </p>
 *
 * @param file_name the name of the dictionary file
 * @param num_codewords number of blocks of the dictionary
 * @param block_size_x horizontal size of the block
 * @param block_size_y vertical size of the block
 *
 */
void
load_dictionary (char *file_name, int *num_codewords, int *block_size_x,
		 int *block_size_y)
{
  int i, j;
  FILE *pointf_dic;

  pointf_dic = fopen (file_name, "r");
  if (pointf_dic == NULL)
    {
      fprintf (stderr, "Impossivel abrir dicionario: %s\n\n", file_name);
      exit (1);
    }

  if (fscanf (pointf_dic, "%d\n", num_codewords) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }
  if (fscanf (pointf_dic, "%d\n", block_size_x) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }
  if (fscanf (pointf_dic, "%d\n", block_size_y) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }

  printf ("\n-----------------------------------------------------");
  printf ("\n Carregou dicionario %s", file_name);
  printf ("\n %d blocos de %dx%d pixels",
	  *num_codewords, *block_size_y, *block_size_x);
  printf ("\n-----------------------------------------------------");
  fflush (stdout);

  G_dic = int_vector (*num_codewords, *block_size_y * (*block_size_x));

  for (i = 0; i < *num_codewords; i++)
    {
      for (j = 0; j < *block_size_x * (*block_size_y); j++)
	{
	  if (fscanf (pointf_dic, "%d\t",
		      &G_dic[i * (*block_size_x * (*block_size_y)) + j]) ==
	      EOF)
	    {
	      printf ("\nEOF ERROR\n");

	    }

	}
      //(void) fscanf(pointf_dic, "\n");
      if (fscanf (pointf_dic, "\n") == EOF)
	{
	  printf ("\nEOF ERROR\n");

	}
    }

  fclose (pointf_dic);
}

/**
 * <p> Calculate the square error between a vector and a training set of the codebook vector. </p>
 *
 * @param index_dic index of the dictionary row
 * @param block_size size of the block (horizontal coordinate * vertical coordinate)
 * @param original_block the current block
 * @return the square error value
 */
float
quad_err (int index_dic, int block_size, int *original_block)
{
  int i;
  float tmp = 0;

  for (i = 0; i < block_size; i++)
    {
      tmp +=
	((G_dic[index_dic * block_size + i] -
	  original_block[i]) * (G_dic[index_dic * block_size + i] -
				original_block[i]));
    }
  return tmp;
}

/**
 *  <p> Reads the information of a pgm file to calculate the horizontal and vertical size.</p>
 *
 * @param ysize image vertical dimension
 * @param xsize image horizontal dimensio
 * @param file_name file name of the image that will be coded
 */
void
read_header_pgm (int *ysize, int *xsize, char *file_name)
{
  FILE *pointf;
  char dummy[12];
  char aux;
  int read = 0;

  pointf = fopen (file_name, "r");
  if (pointf == NULL)
    {
      fprintf (stderr, "\nCannot open input file: %s\n", file_name);
      exit (1);
    }

  if (fscanf (pointf, "%s\n", dummy) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }


  //*******************************
  do
    {
      aux = getc (pointf);
      //fflush(stdout);
      if (aux == '#')
	{
	  do
	    {
	      aux = getc (pointf);
	    }
	  while ((aux != '\n') && (aux != EOF));


	}
      else
	{
	  ungetc (aux, pointf);
	  read = 1;
	}
    }
  while (read == 0);

  if (fscanf (pointf, " %d %d", xsize, ysize) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }
  //********************************

  fclose (pointf);		/* closes file */
}

/**
 *
 * <p> Copy the image to memory </p>
 *
 * @param pelimg vector where the image will be saved
 * @param ysize image vertical dimension
 * @param xsize image horizontal dimensio
 * @param file_name file name of the image that will be coded
 */
void
read_file_pgm (int **pelimg, int *ysize, int *xsize, char *file_name)
{
  int i, j;
  FILE *pointf;
  char dummy[15];
  int aux[1];
  char aux1;
  int read = 0;


  pointf = fopen (file_name, "r");
  if (pointf == NULL)
    {
      fprintf (stderr, "\nCannot open input file: %s\n", file_name);
      exit (1);
    }

  if (fscanf (pointf, "%s\n", dummy) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }

  //*******************************
  do
    {
      aux1 = getc (pointf);
      //fflush(stdout);
      if (aux1 == '#')
	{
	  do
	    {
	      aux1 = getc (pointf);
	    }
	  while ((aux1 != '\n') && (aux1 != EOF));
	}
      else
	{
	  ungetc (aux1, pointf);
	  read = 1;
	}
    }
  while (read == 0);



  if (fscanf (pointf, " %d %d", xsize, ysize) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }
  //********************************

  if (fscanf (pointf, "%d\n", aux) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }

  for (i = 0; i < *ysize; i++)
    {
      for (j = 0; j < *xsize; j++)
	{
	  pelimg[i][j] = (unsigned char) fgetc (pointf);
	}
    }

  fclose (pointf);		/* closes file */
}

/**
 *
 * <p> Copy the image to memory </p>
 *
 * @param pelimg vector where the image will be saved
 * @param ysize image vertical dimension
 * @param xsize image horizontal dimensio
 * @param file_name file name of the image that will be coded
 */
void
v_read_file_pgm (int *pelimg, int *ysize, int *xsize, char *file_name)
{
  int i, j;
  FILE *pointf;
  char dummy[15];
  int aux[1];
  char aux1;
  int read = 0;


  pointf = fopen (file_name, "r");
  if (pointf == NULL)
    {
      fprintf (stderr, "\nCannot open input file: %s\n", file_name);
      exit (1);
    }

  if (fscanf (pointf, "%s\n", dummy) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }

  //*******************************
  do
    {
      aux1 = getc (pointf);
      //fflush(stdout);
      if (aux1 == '#')
	{
	  do
	    {
	      aux1 = getc (pointf);
	    }
	  while ((aux1 != '\n') && (aux1 != EOF));
	}
      else
	{
	  ungetc (aux1, pointf);
	  read = 1;
	}
    }
  while (read == 0);



  if (fscanf (pointf, " %d %d", xsize, ysize) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }
  //********************************

  if (fscanf (pointf, "%d\n", aux) == EOF)
    {
      printf ("\nEOF ERROR\n");

    }

  for (i = 0; i < *ysize; i++)
    {
      for (j = 0; j < *xsize; j++)
	{
	  pelimg[i * (*xsize) + j] = (unsigned char) fgetc (pointf);
	}
    }

  fclose (pointf);		/* closes file */
}

/**
 * <p> Allocates memory for a matrix of variables of type int. </p>
 *
 * @param nr number of rows
 * @param nc number of columns
 * @return a pointer to a int matrix (int **)
 */
int **
int_matrix (int nr, int nc)
{
  int i;
  int **m;

  m = (int **) malloc ((unsigned) (nr) * sizeof (int *));
  if (!m)
    {
      printf ("int_matrix() - allocation failure 1 \n");
      exit (1);
    }

  for (i = 0; i < nr; i++)
    {
      m[i] = (int *) malloc ((unsigned) (nc) * sizeof (int));
      if (!m[i])
	{
	  printf ("int_matrix() - allocation failure 2 \n");
	  exit (1);
	}
    }

  return m;
}

/**
 * <p> Allocates memory for a matrix of variables of type float. </p>
 *
 * @param nr number of rows
 * @param nc number of columns
 * @return a pointer to a int matrix (float **)
 */
float **
floatmatrix (int nr, int nc)
{
  int i;
  float **m;

  m = (float **) malloc (nr * nc * sizeof (float *));
  if (!m)
    {
      printf ("floatmatrix() - allocation failure 1 \n");
      exit (1);
    }

  for (i = 0; i < nr; i++)
    {
      m[i] = (float *) calloc (nc, sizeof (float));
      if (!m[i])
	{
	  printf ("floatmatrix() - allocation failure 2 \n");
	  exit (1);
	}
    }

  return m;
}

/**
 * <p> Allocates memory for a vector of variables of type int. </p>
 *
 * @param nr number of rows
 * @param nc number of columns
 * @return a pointer to a int vector (int *)
 */
int *
int_vector (int nr, int nc)
{
  int *v;

  v = (int *) malloc ((unsigned) (nr * nc) * sizeof (int *));
  if (!v)
    {
      printf ("int_vector() - allocation failure 1 \n");
      exit (1);
    }

  return v;
}

/**
 * <p> This function will sorte all the pixels of each block to be side by side. </p>
 *
 * @param v_pgm vector with the image
 * @param v_sort_pgm vector with the image blocks sorted (all pixels of the block side by side)
 * @param block_x horizontal size of the block
 * @param block_y vertical size of the block
 * @param xsize horizontal size of the image
 * @param ysize vertical size of the image
 */
void
sort_pgm_blocks (int *v_pgm, int *v_sort_pgm, int block_x, int block_y,
		 int xsize, int ysize)
{
  int count = 0, i, i1, j, j1;

  for (i = 0; i < ysize; i += block_y)
    {
      for (j = 0; j < xsize; j += block_x)
	{

	  //Para todos os blocos
	  for (i1 = 0; i1 < block_y; i1++)
	    {
	      for (j1 = 0; j1 < block_x; j1++)
		{

		  v_sort_pgm[count] = v_pgm[(i + i1) * xsize + (j + j1)];
		  count++;
		}
	    }
	}
    }
}

/**
 * <p> Calculate the Peak Signal Noise Ratio </p>
 *
 * @param origblk
 * @param cmpblk
 * @param nline
 * @param npixel
 * @return
 */
double
calculate_psnr (int **origblk, int **cmpblk, int nline, int npixel)
{
  int i, j;
  double psnr;

  psnr = 0.;
  for (j = 0; j < nline; j++)
    for (i = 0; i < npixel; i++)
      psnr +=
	((double) *(origblk[j] + i) -
	 (double) *(cmpblk[j] + i)) * ((double) *(origblk[j] + i) -
				       (double) *(cmpblk[j] + i));

  if (psnr == 0.0)
    psnr = 0.000000000000000000000000000000000000000000000000000001;
  psnr =
    10.0 * log10 ((double) (RANGEY * RANGEY) /
		  (psnr / (double) (nline * npixel)));

  return psnr;
}

/**
 * <p> Mean Squared Error </p>
 *
 * @param origblk
 * @param cmpblk
 * @param nline
 * @param npixel
 * @return
 */
double
calculate_mse (int **origblk, int **cmpblk, int nline, int npixel)
{
  int i, j;
  long cnt = 0;
  double mse;

  mse = 0.;
  for (j = 0; j < nline; j++)
    for (i = 0; i < npixel; i++)
      {
	mse +=
	  ((double) *(origblk[j] + i) -
	   (double) *(cmpblk[j] + i)) * ((double) *(origblk[j] + i) -
					 (double) *(cmpblk[j] + i));
	cnt++;
      }

  return (mse / cnt);
}

/**
 * <p> Output a bit </p>
 *
 * @param bit
 * @param output_file
 * @param buffer
 * @param bits_to_go
 * @param bits_count
 */
void
output_bit (int bit, FILE * output_file, int *buffer, int *bits_to_go,
	    long *bits_count)
{
  *buffer >>= 1;		/* Put bit in top of buffer  */
  if (bit)
    *buffer |= 0x80;
  *bits_to_go -= 1;
  *bits_count += 1;
  if (*bits_to_go == 0)		/* Output buffer if it is     */
    {				/* now full                                        */
      putc (*buffer, output_file);
      /*       printf("  %x\n", buffer); */
      *bits_to_go = 8;
    }
}

/**
 * <p> Flush out the last bits </p>
 *
 * @param output_file
 * @param buffer
 * @param bits_to_go
 */
void
done_outputing_bits (FILE * output_file, int *buffer, int *bits_to_go)
{
  putc (*buffer >> *bits_to_go, output_file);
  //fprintf(stderr," Total Bits: %d bits\n",Bits_Count);
}

/**
 * <p> Write output to luminance file - PGM format </p>
 *
 * @param im_matrix
 * @param nline
 * @param npixel
 * @param filename
 */
void
write_f_pgm (int **im_matrix, int nline, int npixel, char *filename)
{
  int i;
  int pointfo;
  char header_pgm[20];
  int npixel_orig, nline_orig;
	ssize_t bytes_written;

  npixel_orig = npixel;
  nline_orig = nline;

  if ((pointfo = creat (filename, PERMS)) == -1)	/* creates file */
    {
      fprintf (stderr, "Write_f_PGM - cannot create PGM file -> %s",
	       filename);
      exit (1);
    }

  /* Defines the header of file - type pgm */
  *(header_pgm + 0) = 'P';
  *(header_pgm + 1) = '5';
  *(header_pgm + 2) = '\n';

  i = 3;
  if (npixel >= 1000)
    {
      *(header_pgm + i) = (unsigned char) (npixel / 1000 + 48);
      npixel = npixel % 1000;
      i++;
    }
  *(header_pgm + i) = (unsigned char) (npixel / 100 + 48);
  i++;
  *(header_pgm + i) = (unsigned char) (npixel % 100) / 10 + 48;
  i++;
  *(header_pgm + i) = (unsigned char) ((npixel % 100) % 10 + 48);
  i++;

  *(header_pgm + i) = ' ';
  i++;

  if (nline >= 1000)
    {
      *(header_pgm + i) = (unsigned char) (nline / 1000 + 48);
      nline = nline % 1000;
      i++;
    }
  *(header_pgm + i) = (unsigned char) (nline / 100 + 48);
  i++;
  *(header_pgm + i) = (unsigned char) ((nline % 100) / 10 + 48);
  i++;
  *(header_pgm + i) = (unsigned char) ((nline % 100) % 10 + 48);
  i++;

  *(header_pgm + i) = '\n';
  i++;

  *(header_pgm + i) = (unsigned char) (RANGE_LUMINANCE / 100 + 48);
  i++;
  *(header_pgm + i) = (unsigned char) ((RANGE_LUMINANCE % 100) / 10 + 48);
  i++;
  *(header_pgm + i) = (unsigned char) ((RANGE_LUMINANCE % 100) % 10 + 48);
  i++;

  *(header_pgm + i) = '\n';
  i++;

  bytes_written=write (pointfo, (char *) header_pgm, i);

  unsigned char **image_tmp;
  int j;
  image_tmp = ucmatrix (0, nline, 0, npixel);

  for (i = 0; i < nline; i++)
    for (j = 0; j < npixel; j++)
      image_tmp[i][j] = (unsigned char) im_matrix[i][j];

  for (i = 0; i < (nline_orig); i++) {
    bytes_written=write (pointfo, image_tmp[i], npixel_orig);
}
  close (pointfo);		/* closes file */
}

/**
 * <p> Allocates memory for a matrix of variables of type unsigned char </p>
 *
 * @param nrl number of last row
 * @param nrh number of first row
 * @param ncl number of last column
 * @param nch number of first column
 * @return a pointer to a unsigned char matrix (unsigned char **)
 */
unsigned char **
ucmatrix (int nrl, int nrh, int ncl, int nch)
{
  int i;
  unsigned char **m;

  m =
    (unsigned char **) malloc ((unsigned) (nrh - nrl + 1) * sizeof (char *));
  if (!m)
    {
      printf ("ucmatrix() - allocation failure 1 \n");
      exit (1);
    }
  m -= nrl;

  for (i = nrl; i <= nrh; i++)
    {
      m[i] =
	(unsigned char *) malloc ((unsigned) (nch - ncl + 1) * sizeof (char));
      if (!m[i])
	{
	  printf ("ucmatrix() - allocation failure 2 \n");
	  exit (1);
	}
      m[i] -= ncl;
    }
  return m;
}
